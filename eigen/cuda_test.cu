#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <Eigen/Dense>

__global__
void mult_vect(int n, float *x, float *y)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for (int i = index; i < n; i+=stride)
      y[i] = x[i] * y[i];
}

int main(void)
{
  int n = 1<<10; 

  float *x, *y;

  hipMallocManaged(&x, n*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  for (int i = 0; i < n; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  int blockSize = 256;
  int numBlocks = (n + blockSize - 1) / blockSize;

  mult_vect<<<numBlocks, blockSize>>>(n, x, y);

  hipDeviceSynchronize();
  
  // float maxError = 0.0f;
  // for (int i = 0; i < n; i++)
  //   maxError = fmax(maxError, fabs(y[i]-3.0f));
  // std::cout << "error rate: " << maxError << std::endl;

  hipFree(x);
  hipFree(y);

  return 0;
}